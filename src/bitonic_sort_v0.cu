#include "hip/hip_runtime.h"
#include "../inc/bitonic_sort.cuh"

__global__ void bitonic_kernel_v0(int* data, int length, int stage, int step) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threads operate within valid range
    if (idx >= length) return;

    // Find the partner index
    int partner = idx ^ (1 << step);

    // Ensure valid partner index
    if (idx >= partner || partner >= length) return;

    // // Determine if the exchange should be in ascending order
    // bool ascending = (idx & (1 << (stage + 1))) == 0;

    BITONIC_COMPARE_AND_SWAP(idx, idx, partner, stage, data)
}

void bitonic_sort_v0(IntArray& array) {
    int* d_data;
    size_t size = array.length * sizeof(int);

    // Allocate memory on the device
    hipMalloc(&d_data, size);
    hipMemcpy(d_data, array.data, size, hipMemcpyHostToDevice);

    int num_blocks = (array.length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int stages     = __builtin_ctz(array.length); // __builtin_ctz gets log2(length)

    // Launch the Bitonic Sort
    for (int stage = 0; stage < stages; stage++) {
        for (int step = stage; step >= 0; step--) {
            bitonic_kernel_v0<<<num_blocks, THREADS_PER_BLOCK>>>(d_data, array.length, stage, step);

            #ifdef DEBUG
            // Optional kernel error-checking
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                printf("CUDA error: %s\n", hipGetErrorString(err));
                hipFree(d_data);
                return;
            }
            #endif
            
            hipDeviceSynchronize();
        }
    }

    // Copy the sorted data back to the host
    hipMemcpy(array.data, d_data, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_data);
}
